// Homework_5
// Problem_4 
// change the array size to 8000. Check if answer to problem 3 still works.

// RUN as
// nvcc prob4.cu
// ./a.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int sectors = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = sectors; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

//add kernel function to add i to a[i]
__global__
void addIValue(int *arr, int size){
  int sectors = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = sectors; i < size; i+= increment){ 
         arr[i] += i; 
      }
}

//loop
void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}

// it prints out message of running
int main(void){
printf("Homework#5\nProblem 4:Change the array size to 8000. Check if answer to problem 3 still works\n---Successfully initiated---\n---Check the code---");

//here declare int array
int size = 8000;
int *array;
int GPU = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int blocks = (size + GPU - 1) / GPU;
initialize<<<blocks, GPU>>>(array, size);

//here add value of i to array 
addIValue<<<blocks, GPU>>>(array, size);
hipDeviceSynchronize();

print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}

#include "hip/hip_runtime.h"
// Problem 1
// Write a CUDA program to initialize an array of size 32 to all zeros in parallel.
// Homework_5

// RUN by
// nvcc prob1.cu
// ./a.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//We use kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = index; i < size; i += increment){ 
           arr[i] = 0; 
        }
}
//loop
void print(int *ar, int size){
  printf("\n");a
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework#5\nProblem 1: Initialized an array of size 32 to all zeros in parallel\n---Successfully initiated---\n---Check the Code---");

//here we declare int array
int size = 32;
int *array;
int GPU = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int sectors = (size + GPU - 1) / GPU;
initialize<<<sectors, GPU>>>(array, size);

// prints
print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}

// Homework_5
// Problem_3 
// create another kernel that adds i to array[i]

// RUN as:
// nvcc prob3.cu
// ./a.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int indexing = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 
	//loop for index then increment
  for (int i = indexing; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

//Kernel function to add i to a[i]
__global__
void add_I_Value(int *arr, int size){
  int indexing = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = indexing; i < size; i+= increment){ 
         arr[i] += i; 
      }
}

//loop

void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework#5\nProblem 3: create another kernel that adds i to array[i]\n---Successfully initiated---\n---Check the code---");

//here we declare int array
int size = 1024;
int *array;
int gpuThread = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int sectors = (size + gpuThread - 1) / gpuThread;
initialize<<<sectors, gpuThread>>>(array, size);

//here we add value of i to array 
add_I_Value<<<sectors, gpuThread>>>(array, size);
hipDeviceSynchronize();

//prints the array and takes the size
print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}

// Homework 5 
// Problem_2
// Change the array size to 1024

// RUN as:
// nvcc prob2.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//This is kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int indexing = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = indexing; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

// loop
void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework#4\nProblem 2: Change the array size to 1024 in Problem_1\n---Successfully initiated---\n---Check the code---");

//we declare int array
int size = 1024;
int *array;
int GPU = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int sectors = (size + GPU - 1) / GPU;
initialize<<<sectors, GPU>>>(array, size);


//prints
print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}

// execute by typing nvcc que1.cu
// ./a.out

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 32

__global__ void initArray(int *arr)
{
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    arr[tidx] = tidx;
}
__global__ void square (int *matrix,  int *result, int matrixsize) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int ii = id / matrixsize;
    int jj = id % matrixsize;
    int index = ii * matrixsize + jj;

    for (int kk = 0; kk < matrixsize; ++kk) {
        int ix = ii * matrixsize + kk;
        int jx = kk * matrixsize + jj;
        int r = matrix[ix] * matrix[jx];
        printf("Mresult_arr[%d] = %d\n", index, r);
        printf("ix = %d; jx = ;\n", ix, jx);

        result[index] += kk;
    }
}

int main()
{
    int *arr; 
    int *result_arr; 
    int *d_arr; 
    int *d_result_arr; 
    int raw_size = (N * 2);
    int size = raw_size * sizeof(int); 
    arr = (int *)malloc(size);
    result_arr = (int *)malloc(size);
    hipMalloc((void **)&d_arr, size);
    hipMalloc((void **)&d_result_arr, size);

    initArray<<<raw_size,1>>>(d_arr);
    square<<<raw_size,1>>>(d_arr, d_result_arr, raw_size);
    hipMemcpy(result_arr, d_result_arr, size, hipMemcpyDeviceToHost);
    free(arr);
    hipFree(d_arr);
    return 0;
}

